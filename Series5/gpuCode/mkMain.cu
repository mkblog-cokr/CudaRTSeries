#include "hip/hip_runtime.h"
#include <fstream>
#include "mkRay.h"
#include <time.h>
#include "mkSphere.h"
#include "mkHitablelist.h"
#include <float.h>

using namespace std;

//MK: FB 사이즈
int nx = 1200;
int ny = 600;

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
//MK: #val은 val 전체를 String으로 Return 함 (출처 3)
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

//MK: Error 위치를 파악하기 위해서 사용
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        cerr << "MK: CUDA ERROR = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

//MK: (코드 4-2) Sphere 2개를 추가하는 코드
//MK: 1개의 Thread만 연산을 수행할 수 있도록 if문을 추가함
__global__ void mkCreateWorld(hitable **dList, hitable **dWorld){
    if(threadIdx.x == 0 && blockIdx.x == 0){
        *(dList) = new sphere(vec3(0, 0, -1), 0.5);
	*(dList + 1) = new sphere(vec3(0, -100.5, -1), 100);
	*dWorld = new hitableList(dList, 2);
    }
}

//MK: (코드 4-7) 여러개의 Sphere Hit 여부를 판단하여 색상을 결정하도록 코드 변경
__device__ vec3 color(const ray &r, hitable **dWorld){
    hitRecord rec;
    vec3 ret = vec3(0, 0, 0);
    if((*dWorld)->hit(r, 0.0, FLT_MAX, rec)){
	ret = 0.5f * vec3(rec.normal.x() + 1.0f, rec.normal.y() + 1.0f, rec.normal.z() + 1.0f);
    }	
    else{
	vec3 unitDirection = unitVector(r.direction());
    	float t = 0.5f * (unitDirection.y() + 1.0f);
    	ret = (1.0 - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
    }
    return ret;
}

//MK: (코드 4-6) 여러개의 Sphere의 색상을 결정하기 위해서 코드 변경
__global__ void mkRender(vec3 *fb, int max_x, int max_y, vec3 lowerLeftCorner, vec3 horizontal, vec3 vertical, vec3 origin, hitable **dWorld) {
    //MK: Pixel 위치 계산을 위해 ThreadId, BlockId를 사용함
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    //MK: 계산된 Pixel 위치가 FB사이즈 보다 크면 연산을 수행하지 않음
    if((i >= max_x) || (j >= max_y)){
        return;
    }

    //MK: FB Pixel 값 계산
    int pixel_index = j*max_x + i;
    float u = float(i)/float(max_x);
    float v = float(j)/float(max_y);
    ray r(origin, lowerLeftCorner + u*horizontal + v*vertical);
    fb[pixel_index] = color(r, dWorld);
}

//MK: (코드 4-4) mkCreateWorld에서 생성한 클래스 제거
__global__ void mkFreeWorld(hitable **dList, hitable **dWorld){
    if(threadIdx.x == 0 && blockIdx.x == 0){
        delete *(dList);
	delete *(dList + 1);
	delete *dWorld;
    }
}

int main() {
    //MK: Thread Block 사이즈
    int tx = 8;
    int ty = 8;

    cout << "MK: Rendering a " << nx << "x" << ny << " Image ";
    cout << "MK: in " << tx << "x" << ty << " Thread Blocks.\n";

    clock_t start, stop;
    start = clock();

    int num_pixels = nx*ny;
    size_t fb_size = 3*num_pixels*sizeof(float);

    //MK: FB 메모리 할당 (hipMallocManaged 는 Unitifed Memory를 사용 할 수 있도록 함)
    //MK: 필요에 따라 CPU/GPU에서 GPU/CPU로 데이터를 복사함
    vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    //MK: (코드 4-1) Sphere을 여러개 추가하기 위해서 메모리 할당을 진행
    hitable **dList;
    hitable **dWorld;
    checkCudaErrors(hipMalloc((void **) &dList, 2 * sizeof(hitable *)));
    checkCudaErrors(hipMalloc((void **) &dWorld, sizeof(hitable *)));
	
    //MK: (코드 4-3) Sphere를 생성하는 함수를 호출함
    mkCreateWorld<<<1, 1>>>(dList, dWorld);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    //MK: GPU (CUDA) 연산을 위해서 Thread Block, Grid 사이즈 결정
    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);
    //MK: CUDA 함수 호출
    mkRender<<<blocks, threads>>>(fb, nx, ny,
					vec3(-2.0, -1.0, -1.0),
					vec3(4.0, 0.0, 0.0),
					vec3(0.0, 2.0, 0.0),
					vec3(0.0, 0.0, 0.0),
					dWorld);
    checkCudaErrors(hipGetLastError());
    //MK: CUDA 연산이 완료되길 기다림
    checkCudaErrors(hipDeviceSynchronize());
    //MK: 연산 시간과 끝 부분을 계산하여서 연산 시간을 측정함 

    //MK: CPU 코드와 동일하게 결과를 파일에 작성
    string fileName = "Ch5_gpu.ppm";
    ofstream writeFile(fileName.data());
    if(writeFile.is_open()){
	writeFile.flush();
	writeFile << "P3\n" << nx << " " << ny << "\n255\n";
    	for (int j = ny-1; j >= 0; j--) {
            for (int i = 0; i < nx; i++) {
                size_t pixel_index = j*nx + i;
            	int ir = int(255.99 * fb[pixel_index].r());
            	int ig = int(255.99 * fb[pixel_index].g());
            	int ib = int(255.99 * fb[pixel_index].b());
            	writeFile  << ir << " " << ig << " " << ib << "\n";
            }
    	}
	writeFile.close();
    }

    //MK: (코드 4-5) 사용한 메모리를 제거함
    mkFreeWorld<<<1, 1>>>(dList, dWorld);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(dList));
    checkCudaErrors(hipFree(dWorld));
    checkCudaErrors(hipFree(fb));
	
    //MK: 연산 시간과 끝 부분을 계산하여서 연산 시간을 측정함 
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    cout << "MK: GPU (CUDA) Took " << timer_seconds << " Seconds.\n";

    return 0;
}