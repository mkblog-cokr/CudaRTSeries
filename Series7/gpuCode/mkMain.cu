#include "hip/hip_runtime.h"
#include <fstream>
#include "mkRay.h"
#include <time.h>
#include "mkSphere.h"
#include "mkHitablelist.h"
#include <float.h>
#include <hiprand/hiprand_kernel.h>
#include "mkCamera.h"

using namespace std;

//MK: FB 사이즈
int nx = 1200;
int ny = 600;
int ns = 100;

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
//MK: #val은 val 전체를 String으로 Return 함 (출처 3)
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

//MK: Error 위치를 파악하기 위해서 사용
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        cerr << "MK: CUDA ERROR = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void mkCreateWorld(hitable **dList, hitable **dWorld, camera **dCamera){
    if(threadIdx.x == 0 && blockIdx.x == 0){
        *(dList) = new sphere(vec3(0, 0, -1), 0.5);
    *(dList + 1) = new sphere(vec3(0, -100.5, -1), 100);
    *dWorld = new hitableList(dList, 2);
        *dCamera = new camera();
    }
}

//MK: (코드 1-2) Random 하게 Vector을 생성하기 위한 코드
#define RANDVEC3 vec3(hiprand_uniform(localRandState), hiprand_uniform(localRandState), hiprand_uniform(localRandState))

//MK: (코드 1-3) Unit Sphere의 Random한 포인터를 생성하기 위한 코드
__device__ vec3 randomInUnitSphere(hiprandState *localRandState){
    vec3 p;
    do{
    p = 2.0f * RANDVEC3 - vec3(1.0, 1.0, 1.0);
    } while (p.squared_length() >= 1.0f);
    return p;
}

//MK: (코드 1-1) Recurisve 함수를 Loop을 사용하도록 변경함
__device__ vec3 color(const ray &r, hitable **dWorld, hiprandState *localRandState){
    ray curRay = r;
    float curAttenuation = 1.0f;
    for(int i = 0; i < 50; i++){
    hitRecord rec;
    if((*dWorld)->hit(curRay, 0.001f, FLT_MAX, rec)){
            vec3 target = rec.p + rec.normal + randomInUnitSphere(localRandState);
        curAttenuation *= 0.5f;
        curRay = ray(rec.p, target-rec.p);
    }
    else{
        vec3 unitDirection = unitVector(curRay.direction());
        float t = 0.5f * (unitDirection.y() + 1.0f);
        vec3 c = (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
        return curAttenuation * c;
    }
    }
    return vec3(0.0, 0.0, 0.0);
}

__global__ void mkRender(vec3 *fb, int max_x, int max_y, int num_sample, camera **cam, hitable **dWorld) {
    //MK: Pixel 위치 계산을 위해 ThreadId, BlockId를 사용함
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    //MK: 계산된 Pixel 위치가 FB사이즈 보다 크면 연산을 수행하지 않음
    if((i >= max_x) || (j >= max_y)){
        return;
    }

    //MK: FB Pixel 값 계산
    int pixel_index = j*max_x + i;
    
    hiprandState rand_state;
    //hiprand_init(1984, pixel_index, 0, &rand_state);
    hiprand_init(pixel_index, 0, 0, &rand_state);
    vec3 col(0, 0, 0);
    for(int s = 0; s < num_sample; s++){
    float u = float(i + hiprand_uniform(&rand_state))/float(max_x);
        float v = float(j + hiprand_uniform(&rand_state))/float(max_y);
    ray r = (*cam)->get_ray(u, v);
    col += color(r, dWorld, &rand_state);
    }
    fb[pixel_index] = col/float(num_sample);
}

__global__ void mkFreeWorld(hitable **dList, hitable **dWorld, camera **dCamera){
    if(threadIdx.x == 0 && blockIdx.x == 0){
        delete *(dList);
    delete *(dList + 1);
    delete *dWorld;
    delete *dCamera;
    }
}

int main() {
    //MK: Thread Block 사이즈
    int tx = 8;
    int ty = 8;

    cout << "MK: Rendering a " << nx << "x" << ny << " Image ";
    cout << "MK: in " << tx << "x" << ty << " Thread Blocks.\n";

    clock_t start, stop;
    start = clock();

    int num_pixels = nx*ny;
    size_t fb_size = 3*num_pixels*sizeof(float);

    //MK: FB 메모리 할당 (hipMallocManaged 는 Unitifed Memory를 사용 할 수 있도록 함)
    //MK: 필요에 따라 CPU/GPU에서 GPU/CPU로 데이터를 복사함
    vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    hitable **dList;
    hitable **dWorld;
    camera **dCamera;
    checkCudaErrors(hipMalloc((void **) &dList, 2 * sizeof(hitable *)));
    checkCudaErrors(hipMalloc((void **) &dWorld, sizeof(hitable *)));
    checkCudaErrors(hipMalloc((void **) &dCamera, sizeof(camera *)));
        
    mkCreateWorld<<<1, 1>>>(dList, dWorld, dCamera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    //MK: GPU (CUDA) 연산을 위해서 Thread Block, Grid 사이즈 결정
    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);
    //MK: CUDA 함수 호출
    mkRender<<<blocks, threads>>>(fb, nx, ny, ns, dCamera, dWorld);
    checkCudaErrors(hipGetLastError());
    //MK: CUDA 연산이 완료되길 기다림
    checkCudaErrors(hipDeviceSynchronize());
    //MK: 연산 시간과 끝 부분을 계산하여서 연산 시간을 측정함 

    //MK: CPU 코드와 동일하게 결과를 파일에 작성
    string fileName = "Ch7_gpu.ppm";
    ofstream writeFile(fileName.data());
    if(writeFile.is_open()){
    writeFile.flush();
    writeFile << "P3\n" << nx << " " << ny << "\n255\n";
        for (int j = ny-1; j >= 0; j--) {
            for (int i = 0; i < nx; i++) {
                size_t pixel_index = j*nx + i;
                int ir = int(255.99 * fb[pixel_index].r());
                int ig = int(255.99 * fb[pixel_index].g());
                int ib = int(255.99 * fb[pixel_index].b());
                writeFile  << ir << " " << ig << " " << ib << "\n";
            }
        }
    writeFile.close();
    }

    mkFreeWorld<<<1, 1>>>(dList, dWorld, dCamera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(dList));
    checkCudaErrors(hipFree(dWorld));
    checkCudaErrors(hipFree(fb));
    checkCudaErrors(hipFree(dCamera));
    
    //MK: 연산 시간과 끝 부분을 계산하여서 연산 시간을 측정함 
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    cout << "MK: GPU (CUDA) Took " << timer_seconds << " Seconds.\n";

    return 0;
}
